
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void write_ids(int* output) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    output[global_id] = global_id * global_id;
}


int main(){
    int N = 8;

    // 1. Reservar memoria en la GPU
    int* device_array;
    hipMalloc(&device_array, N * sizeof(int));

    // 2. Lanzar el kernel
    write_ids<<<2, 4>>>(device_array);  // 2 bloques de 4 = 8 hilos
    hipDeviceSynchronize();

    // 3. Copiar resultado al host
    int host_array[N];
    hipMemcpy(host_array, device_array, N * sizeof(int), hipMemcpyDeviceToHost);

    // 4. Verificar
    for (int i = 0; i < N; ++i) {
        printf("host_array[%d] = %d\n", i, host_array[i]);
    }

    // 5. Liberar
    hipFree(device_array);

}