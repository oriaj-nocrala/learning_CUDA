#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "helper_structs.h"
#include "file_manager.h"
#include "error_checker.h"

#include "cell_structure.cuh"
#include "calculate_density.cuh"
#include "calculate_force.cuh"
#include "integrators.cuh"

#include "visualizer.h"
#include "particlePositionCopy.cu"
#include "triangleBufferCopy.cu"
#include <cuda_gl_interop.h>

#include "src/MarchingCubes/CudaGrid.h"
#include "src/MarchingCubes/MarchingCubes.h"


void initializeWallParticles(std::vector<Particle>&, Parameters&);
void initializeParticles(std::vector<Particle>&, Parameters&);
void placeWallParticles(std::vector<Particle>& particles, Parameters& p, int limit, float3 offset, int3 order);
void initializeDensity(std::vector<float3>& density, Parameters& p);

int main() {
	FileManager file_manager("parameter_files", "params_0.par");
	Parameters params = file_manager.readParams();

	std::vector<Particle> particles;
	initializeWallParticles(particles, params);
	initializeParticles(particles, params);

	std::vector<float3> densities;
	initializeDensity(densities, params);

	// Add particle_num (movable) and dam_particle_num (solid), both types of particles are stored in particles
	size_t total_particle_num = params.movable_particle_num + params.immovable_particle_num;
	std::vector<int> cell_list(params.cell_num, -1);
	std::vector<int> particle_list(total_particle_num, -1);

	/* Allocate memory on device */
	Particle* d_particles;
	float3* d_force_buffer;
	float* d_density_buffer;
	int* d_particle_list, * d_cell_list;
	size_t bytes_vec = sizeof(float) * total_particle_num;
	size_t bytes_vec3 = sizeof(float3) * total_particle_num;
	size_t bytes_struct = sizeof(Particle) * total_particle_num;
	size_t bytes_particle_list = sizeof(int) * total_particle_num;
	size_t bytes_cell_list = sizeof(int) * params.cell_num;

	checkError(hipMalloc((void**)&d_particle_list, bytes_particle_list));
	checkError(hipMalloc((void**)&d_cell_list, bytes_cell_list));
	checkError(hipMalloc((void**)&d_particles, bytes_struct));
	checkError(hipMalloc(&d_force_buffer, bytes_vec3));
	checkError(hipMalloc((void**)&d_density_buffer, bytes_vec));

	/* Copy data to device */
	checkError(hipMemcpy(d_particles, particles.data(), bytes_struct, hipMemcpyHostToDevice));
	checkError(hipMemcpy(d_particle_list, particle_list.data(), bytes_particle_list, hipMemcpyHostToDevice));
	checkError(hipMemcpy(d_cell_list, cell_list.data(), bytes_cell_list, hipMemcpyHostToDevice));
	checkError(hipMemcpy(d_density_buffer, densities.data(), bytes_vec, hipMemcpyHostToDevice));

	/* Marching Cubes init */
	float3 box = params.max_box_bound - params.min_box_bound;
	float3 voxelSpacing = box / 128;
	const uint3 gridSize = make_uint3(128);
	const unsigned int maxNumTriangles = 10000000;
	const float isoValue = 0.1;

	std::cout << "Generating sphere ... ";
	CudaGrid grid = CudaGrid::Sphere(gridSize, voxelSpacing);
	std::cout << "done!" << std::endl;

	MarchingCubes marchingCubes(maxNumTriangles);

	/* Visualization init */
	Visualizer vis(params.draw_number, maxNumTriangles, params.particle_radius, params.min_box_bound.x, params.min_box_bound.y, params.min_box_bound.z,
		params.max_box_bound.x, params.max_box_bound.y, params.max_box_bound.z);

	struct hipGraphicsResource* positionsVBO_CUDA = NULL;
	checkError(hipGraphicsGLRegisterBuffer(&positionsVBO_CUDA, vis.vertexArray, cudaGraphicsMapFlagsWriteDiscard));

	struct hipGraphicsResource* trianlgesVBO_CUDA = NULL;
	checkError(hipGraphicsGLRegisterBuffer(&trianlgesVBO_CUDA, vis.triangleArray, cudaGraphicsMapFlagsWriteDiscard));

	if (params.integrator == Integrator::Leapfrog) {
		/* Initialize cell list and particle list */
		assign_to_cells << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list,
			total_particle_num, params.immovable_particle_num, params.cell_dims, params.min_box_bound, params.h_inv);
		checkError(hipPeekAtLastError());
		checkError(hipDeviceSynchronize());

		/* Calculate densities */
		calculate_density << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list, d_density_buffer,
			params.cell_dims, params.min_box_bound, total_particle_num, params.immovable_particle_num, params.h, params.h2, params.h_inv, params.const_poly6, params.mass, params.p0);
		checkError(hipPeekAtLastError());
		checkError(hipDeviceSynchronize());

		/* Calculate forces */
		calculate_force << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list, d_force_buffer, d_density_buffer, params.cell_dims, params.min_box_bound,
			total_particle_num, params.immovable_particle_num, params.h, params.h_inv, params.const_spiky, params.const_visc, params.const_surf, params.mass, params.k, params.e, params.p0, params.s, params.g);
		checkError(hipPeekAtLastError());
		checkError(hipDeviceSynchronize());
	}
	
	std::cout << "Simulation started" << std::endl;
	std::cout << params.spawn_dist << std::endl;
	while (!glfwWindowShouldClose(vis.window)) {
		if (vis.runSimulation) {
			// Start time measurement
			std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

			if (params.integrator == Integrator::Leapfrog) {
				/* Integrate position and velocity */
				leapfrog_pre_integration << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_force_buffer, params.mass_inv, params.time_step,
					total_particle_num, params.immovable_particle_num, params.dam_particle_num, params.min_box_bound, params.max_box_bound, params.damping);
			}

			/* Set all entries of cell list to -1 */
			reset_cell_list << <params.thread_groups_cell, params.threads_per_group >> > (d_cell_list, params.cell_num);
			checkError(hipPeekAtLastError());
			checkError(hipDeviceSynchronize());

			/* Initialize cell list and particle list */
			assign_to_cells << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list,
				total_particle_num, params.immovable_particle_num, params.cell_dims, params.min_box_bound, params.h_inv);
			checkError(hipPeekAtLastError());
			checkError(hipDeviceSynchronize());

			/* Calculate densities */
			calculate_density << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list, d_density_buffer,
				params.cell_dims, params.min_box_bound, total_particle_num, params.immovable_particle_num, params.h, params.h2, params.h_inv, params.const_poly6, params.mass, params.p0);
			checkError(hipPeekAtLastError());
			checkError(hipDeviceSynchronize());

			/* Calculate forces */
			calculate_force << <params.thread_groups_part, params.threads_per_group >> > (d_particles, d_cell_list, d_particle_list, d_force_buffer, d_density_buffer, params.cell_dims, params.min_box_bound,
				total_particle_num, params.immovable_particle_num, params.h, params.h_inv, params.const_spiky, params.const_visc, params.const_surf, params.mass, params.k, params.e, params.p0, params.s, params.g);
			checkError(hipPeekAtLastError());
			checkError(hipDeviceSynchronize());

			/* Set forces of dam particles */
			if (vis.openDam) {
				set_dam_force << < params.thread_groups_part, params.threads_per_group >>> (d_particles, d_force_buffer, total_particle_num, params.immovable_particle_num, params.dam_particle_num);
				checkError(hipPeekAtLastError());
				checkError(hipDeviceSynchronize());
			}

			if (params.integrator == Integrator::Leapfrog) {
				/* Integrate new positions and velocities */
				leapfrog_post_integration << <params.thread_groups_part, params.threads_per_group >> >
					(d_particles, d_force_buffer, params.mass_inv, params.time_step, total_particle_num, params.immovable_particle_num, params.dam_particle_num, params.min_box_bound, params.max_box_bound, params.damping);
			}
			else {
				/* Integrate new positions and velocities */
				integrate_symplectic_euler << <params.thread_groups_part, params.threads_per_group >> >
					(d_particles, d_force_buffer, params.time_step, total_particle_num, params.immovable_particle_num, params.dam_particle_num, params.min_box_bound, params.max_box_bound, params.damping);
				checkError(hipPeekAtLastError());
				checkError(hipDeviceSynchronize());
			}

			// Stop time measurement
			std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
			//std::cout << "Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "ms" << std::endl;

			/* Visualization update */
			float* vertexPointer;
			size_t numBytes;

			if (vis.marchingCubes) {
				/* Marching Cubes */
				// update grid
				int gridSizeN = gridSize.x * gridSize.y * gridSize.z;
				fill_grid<<<gridSizeN, params.threads_per_group>>> (grid, d_particles, gridSizeN, d_cell_list, d_particle_list, d_density_buffer,
					params.cell_dims, params.min_box_bound, params.immovable_particle_num, params.h, params.h2, params.h_inv, params.const_poly6, params.mass, params.p0);
				checkError(hipPeekAtLastError());
				checkError(hipDeviceSynchronize());
				update_grid_normals<<<gridSizeN, params.threads_per_group>>> (grid, d_particles, gridSizeN, d_cell_list, d_particle_list, d_density_buffer,
					params.cell_dims, params.min_box_bound, params.draw_number, params.h, params.h2, params.h_inv, params.const_poly6, params.mass, params.p0);
				checkError(hipPeekAtLastError());
				checkError(hipDeviceSynchronize());

				// reset
				unsigned int numTriangles = 0;
				hipMemset(marchingCubes.marchingCubesData.d_numTriangles, 0, sizeof(unsigned int));
				marchingCubes.extractTrianglesGPU(grid, isoValue);
				checkError(hipMemcpy(&numTriangles, marchingCubes.marchingCubesData.d_numTriangles, sizeof(int), hipMemcpyDeviceToHost));
				
				//std::cout << "numTriangles: " << numTriangles << std::endl;

				// Map the buffer to CUDA
				checkError(hipGraphicsMapResources(1, &trianlgesVBO_CUDA));
				checkError(hipGraphicsResourceGetMappedPointer((void **)&vertexPointer, &numBytes, trianlgesVBO_CUDA));
				// Run kernel
				copy_triangles<<<3 * numTriangles, params.threads_per_group>>>((float*)vertexPointer, (float3*)marchingCubes.marchingCubesData.d_triangles, 3 * numTriangles, params.min_box_bound, grid);
				checkError(hipPeekAtLastError());
				checkError(hipDeviceSynchronize());
				// Unmap the buffer
				checkError(hipGraphicsUnmapResources(1, &trianlgesVBO_CUDA));

				vis.drawTriangles(numTriangles*3);
			} else {
				// Map the buffer to CUDA
				checkError(hipGraphicsMapResources(1, &positionsVBO_CUDA));
				checkError(hipGraphicsResourceGetMappedPointer((void**)&vertexPointer, &numBytes, positionsVBO_CUDA));
				// Run kernel
				copy_particle_positions << <params.thread_groups_part, params.threads_per_group >> > ((float*)vertexPointer, d_particles, total_particle_num, params.immovable_particle_num, params.dam_particle_num);
				// Unmap the buffer
				checkError(hipGraphicsUnmapResources(1, &positionsVBO_CUDA));

				vis.draw(params.draw_number);
			}

			// Stop time measurement
			end = std::chrono::steady_clock::now();
			//std::cout << "Time2 = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "ms" << std::endl;

		}
	}
	std::cout << "Simulation finished" << std::endl;

	/* Free memory on device */
	checkError(hipFree(d_particles));
	checkError(hipFree(d_force_buffer));
	checkError(hipFree(d_density_buffer));
	checkError(hipFree(d_particle_list));
	checkError(hipFree(d_cell_list));

	/* Visualization end */
	vis.end();
}

/* Determines position of wall particles, should be called before initializing moveable particles */
void initializeWallParticles(std::vector<Particle>& particles, Parameters& p) {
	// Calculate shift in order to spawn the cubic shape in the center of the box
	// Shift equals half of the length of the cubic shape
	float shift = p.particle_radius;
	int wall_num = 5;
	int limit;
	float3 offset;

	// boundary bottom
	offset = make_float3(p.min_box_bound.x + shift, p.min_box_bound.z + shift, p.min_box_bound.y + shift);
	limit = p.particle_depth_per_dim.z * p.particle_depth_per_dim.x;
	placeWallParticles(particles, p, limit, offset, make_int3(0, 2, 1));

	// boundary in "false" z front
	offset = make_float3(p.min_box_bound.z + shift, p.min_box_bound.y + shift, p.min_box_bound.x + shift);
	limit = p.particle_depth_per_dim.z* p.particle_depth_per_dim.y;
	placeWallParticles(particles, p, limit, offset, make_int3(2, 1, 0));

	// boundary in "false" z back
	offset = make_float3(p.min_box_bound.z + shift, p.min_box_bound.y + shift, p.max_box_bound.x - shift);
	limit = p.particle_depth_per_dim.z * p.particle_depth_per_dim.y;
	placeWallParticles(particles, p, limit, offset, make_int3(2, 1, 0));

	// boundary in "false" x right
	offset = make_float3(p.min_box_bound.x + shift, p.min_box_bound.y + shift, p.max_box_bound.z - shift);
	limit = p.particle_depth_per_dim.x * p.particle_depth_per_dim.y;
	placeWallParticles(particles, p, limit, offset, make_int3(0, 1, 2));

	// boundary in "false" x left
	offset = make_float3(p.min_box_bound.x + shift, p.min_box_bound.y + shift, p.min_box_bound.z + shift);
	limit = p.particle_depth_per_dim.x * p.particle_depth_per_dim.y;
	placeWallParticles(particles, p, limit, offset, make_int3(0, 1, 2));

	// dam in "false" x right
	offset = make_float3(p.min_box_bound.x + shift, p.min_box_bound.y + shift, (p.min_box_bound.z + p.max_box_bound.z) / 2);
	limit = p.particle_depth_per_dim.x * p.particle_depth_per_dim.y;
	placeWallParticles(particles, p, limit, offset, make_int3(0, 1, 2));

}

void placeWallParticles(std::vector<Particle>& particles, Parameters& p, int limit, float3 offset, int3 order) {
	float coordiantes[3];
	int particle_depths[3] = {p.particle_depth_per_dim.x, p.particle_depth_per_dim.y, p.particle_depth_per_dim.z};

	for (int i = 0; i < limit; i++) {

		// Calculate wall shape
		coordiantes[0] = (i % particle_depths[order.x]) * p.boundary_spawn_dist;
		coordiantes[1] = floor((i / particle_depths[order.x])) * p.boundary_spawn_dist;

		// Add offset
		coordiantes[0] += offset.x;
		coordiantes[1] += offset.y;
		coordiantes[2] = offset.z;

		float3 position = make_float3(coordiantes[order.x], coordiantes[order.y], coordiantes[order.z]);
		particles.emplace_back(position, make_float3(0., 0., 0.));
	}
}

/* Spawns particles in a cubic shape */
void initializeParticles(std::vector<Particle>& particles, Parameters& p) {
	// Calculate shift in order to spawn the cubic shape in the center of the box
	// Shift equals half of the length of the cubic shape
	float shift = (p.edge_length * p.spawn_dist) / 2;

	for (int i = 0; i < p.movable_particle_num; i++) {
		
		// Calculate cubic shape
		float x = (i % p.edge_length) * p.spawn_dist;
		float y = ((i / p.edge_length) % p.edge_length) * p.spawn_dist;
		float z = (i / (p.edge_length * p.edge_length)) * p.spawn_dist;

		// Add offsets
		x += p.spawn_offset.x - shift;
		y += p.spawn_offset.y - shift;
		z += p.spawn_offset.z - shift;

		particles.emplace_back(make_float3(x, y, z), make_float3(0., 0., 0.));
	}
}

void initializeDensity(std::vector<float3>& density, Parameters& p) {

	for (int i = 0; i < p.immovable_particle_num; i++) {
		density.emplace_back(make_float3(p.wall_density, p.wall_density, p.wall_density));
	}
}